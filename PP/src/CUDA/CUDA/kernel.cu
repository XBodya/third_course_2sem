﻿#include "hip/hip_runtime.h"
#include "tools.cuh"
#include "base.cuh"
#include <stdio.h>

#ifdef _DEBUG
#include <iostream>
#endif


#ifdef _DEBUG
template<typename T>
void print(Array<T> arr) {
    for (int i = 0; i < arr.len; ++i) {
        std::cout << arr.ptr[i] << ' ';
    }
}
#endif

int main() {
    printDevicesInformation();
#ifdef _DEBUG
    //printf("%d", 1);
    Array<int> arr(10, 8);
    print(arr);
#endif
    return 0;
}